
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>

#include <fstream>
#include <string>


char* readPPM(char* filename, int* width, int* height) {
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h * 3;
    char* pixel_data = new char[size];

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

void writePPM(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P6" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height) * 3;

    file.write(img_data, size);
}

void writePPMGrey(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P5" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height);

    file.write(img_data, size);
}

char* createImage(char* filename, int width, int height)
{
    char* img = (char*)malloc(sizeof(char) * width * height * 3);
    char pixel;

    for (int y = 0; y < height; y++)
    {
        pixel = rand() % 256;
        for (int x = 0; x < width; x++)
        {
            img[3 * (y * width + x) + 0] = pixel;
            img[3 * (y * width + x) + 1] = pixel;
            img[3 * (y * width + x) + 2] = pixel;
        }
    }

    writePPM(filename, img, width, height);
    return img;
}