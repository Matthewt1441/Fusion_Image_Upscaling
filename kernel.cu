#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>

#include <fstream>
#include <string>

#include "helper.cuh"
#include "serial_code.cuh"
#include "naive_cuda.cuh"
#include "ppm_image.cuh"

#include <chrono>

#include <SDL.h>
#undef main
#include <SDL_ttf.h>
#undef main


void Artifact_Detection(unsigned char* mask_img, unsigned char* img_1, unsigned char* img_2, int* width, int* height, int* window_size, float TH)
{
    int num_windows_x = ceil(*width / (float) *window_size);
    int num_windows_y = ceil(*height / (float) *window_size);

    int total_win_pix = *window_size * *window_size;

    float* img1_window = (float*)malloc(sizeof(float) * total_win_pix * 3);
    float* img2_window = (float*)malloc(sizeof(float) * total_win_pix * 3);

    float* difference_window = (float*)malloc(sizeof(float) * *width * *height * 3);
    float* ssim_window = (float*)malloc(sizeof(float) * 3);

    int img_x = 0; 
    int img_y = 0;

    float* metric_img = (float*)malloc(sizeof(float) * total_win_pix * 3);
    float metric_temp;
    //(Metric < TH) ? 0 : 1

    for (int win_y = 0; win_y < num_windows_y; win_y++)
    {
        for (int win_x = 0; win_x < num_windows_x; win_x++)
        {
            //DATA LOADING PHASE
            for (int y = 0; y < *window_size; y++)
            {
                for (int x = 0; x < *window_size; x++)
                {
                    img_x = x + win_x * *window_size;
                    img_y = y + win_y * *window_size;

                    if ((img_y < *height) && (img_x < *width))
                    {
                        for (int chn = 0; chn < CHN_NUM; chn++)
                        {
                            img1_window[(y * *window_size + x) * CHN_NUM + chn] = (float) img_1[(img_y * *width + img_x) * CHN_NUM + chn];
                            img2_window[(y * *window_size + x) * CHN_NUM + chn] = (float) img_2[(img_y * *width + img_x) * CHN_NUM + chn];
                        }
                    }
                    else
                    {
                        for (int chn = 0; chn < CHN_NUM; chn++)
                        {
                            img1_window[(y * *window_size + x) * CHN_NUM + chn] = -1.0;
                            img2_window[(y * *window_size + x) * CHN_NUM + chn] = -1.0;
                        }
                    }
                }
            }

            //Image Difference Phase
            ABS_Difference(difference_window, img1_window, img2_window, window_size, window_size);
            SSIM(ssim_window, img1_window, img2_window, window_size, window_size);

            //Metric Calculations
            for (int y = 0; y < *window_size; y++)
            {
                for (int x = 0; x < *window_size; x++)
                {
                    img_x = x + win_x * *window_size;
                    img_y = y + win_y * *window_size;

                    if ((img_y < *height) && (img_x < *width))
                    {
                        for (int chn = 0; chn < CHN_NUM; chn++)
                        {
                            metric_temp = difference_window[(y * *window_size + x) * CHN_NUM + chn] * ssim_window[chn];
                            mask_img[(img_y * *width + img_x) * CHN_NUM + chn] = (unsigned char) metric_temp;
                            //mask_img/*metric_img*/[(y * *window_size + x) * CHN_NUM + chn]
                        }
                    }
                }
            }
            //Gausian Blur Stage

        }

    }

    //Reminder to free all the stuff
    free(img1_window); free(img2_window);
    free(difference_window);  free(ssim_window);
    free(metric_img);
    
}

int serialExecution()
{
    try
    {
        int* width = (int*)malloc(sizeof(int));
        int* height = (int*)malloc(sizeof(int));
        unsigned char* img;

        int big_width;
        int big_height;
        int window_size = 8;

        int scale = 2;

        if (SDL_Init(SDL_INIT_VIDEO) < 0) {
            printf("SDL initialization failed: %c\n", SDL_GetError());
            return 1;
        }

        // Initialize SDL_ttf
        if (TTF_Init() < 0) {
            printf("SDL_ttf could not initialize! TTF_Error: %s\n", TTF_GetError());
            SDL_Quit();
            return EXIT_FAILURE;
        }

        bool RUNNING = true;
        bool firstImg = true;
        SDL_Window* window;
        SDL_Renderer* renderer;
        SDL_Texture* texture;
        SDL_Event event;
        SDL_PollEvent(&event);


        int const_width;
        int const_height;

        float diff = 0;
        unsigned char* big_img_nn;
        unsigned char* big_img_nn_grey;
        unsigned char* big_img_bic;
        unsigned char* big_img_bic_grey;
        unsigned char* big_img_dif;
        unsigned char* big_img_dif_grey;
        unsigned char* big_img_ssim_grey;


        TTF_Font* Sans = TTF_OpenFont("Sans.ttf", 24);

        SDL_Color White = { 255, 255, 255 };

        char fps_str[50];
        char file_name[50];

        // as TTF_RenderText_Solid could only be used on
        // SDL_Surface then you have to create the surface first
        SDL_Surface* fps_msg;
        SDL_Texture* fps_txt;

        SDL_Rect Message_rect; //create a rect
        Message_rect.x = 5;  //controls the rect's x coordinate 
        Message_rect.y = 5; // controls the rect's y coordinte
        Message_rect.w = 200; // controls the width of the rect
        Message_rect.h = 30; // controls the height of the rect
        int count = 0;


        double frame_cap = 10;
        sprintf(fps_str, "FPS:%.*f", 3, 0.0);

        int max_image = 200;
        int current_img = 1;

        double processing_time = 0;

        while (RUNNING && event.type != SDL_QUIT)
        {
            if (count == frame_cap)
            {
                diff = 1000 * frame_cap / processing_time;
                sprintf(fps_str, "FPS:%.*f", 3, diff);

                count = 0;
                processing_time = 0;
            }

            sprintf(file_name, "./LM_Frame/image%d.ppm", current_img);

            img = (unsigned char*)readPPM(file_name, width, height);

            auto start = std::chrono::high_resolution_clock::now();

            const_width = *width;
            const_height = *height;

            big_width = const_width * scale; big_height = const_height * scale;
            big_img_nn = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height * 3);
            big_img_nn_grey = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height);
            big_img_bic = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height * 3);
            big_img_bic_grey = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height);
            big_img_dif = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height * 3);
            big_img_dif_grey = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height);
            big_img_ssim_grey = (unsigned char*)malloc(sizeof(unsigned char) * (big_width-8) * (big_height-8));
            //unsigned char* big_img_ssim = (unsigned char*)malloc(sizeof(unsigned char) * *big_width * *big_height * 3);

            //printf("Image dimensions: %d x %d\n", *width, *height);
            //printf("Upscale Image dimensions: %d x %d\n", *big_width, *big_height);

            //nearestNeighbors(big_img_nn, big_width, big_height, img, width, height, scale);
            nearestNeighbors(big_img_nn, big_width, big_height, img, const_width, const_height, scale);
            RGB2Greyscale(big_img_nn, big_img_nn_grey, big_width, big_height);
            bicubicInterpolation(big_img_bic, big_width, big_height, img, const_width, const_height, scale);
            RGB2Greyscale(big_img_bic, big_img_bic_grey, big_width, big_height);

            ABS_Difference_Grey(big_img_dif_grey, big_img_nn_grey, big_img_bic_grey, big_width, big_height);
            //ABS_Difference(big_img_dif, big_img_nn, big_img_bic, big_width, big_height);
            //Artifact_Detection(big_img_dif, big_img_nn, big_img_bic, big_width, big_height, window_size, 0.9);
            SSIM_Grey(big_img_ssim_grey, big_img_nn_grey, big_img_bic_grey, big_width, big_height);

            //writePPM("output_NN.ppm", (char*)big_img_nn, big_width, big_height);
            //writePPM("output_BIC.ppm", (char*)big_img_bic, big_width, big_height);
            //writePPM("output_diff.ppm", (char*)big_img_dif, big_width, big_height);

            auto end = std::chrono::high_resolution_clock::now();
            auto dur = end - start;

            processing_time += std::chrono::duration_cast<std::chrono::milliseconds>(dur).count();

            if (firstImg)
            {

                writePPMGrey("output_NN_grey.ppm", (char*)big_img_nn_grey, big_width, big_height);
                writePPMGrey("output_BIC_grey.ppm", (char*)big_img_bic_grey, big_width, big_height);
                writePPMGrey("output_DIFF_grey.ppm", (char*)big_img_dif_grey, big_width, big_height);
                writePPMGrey("output_SSIM_grey.ppm", (char*)big_img_ssim_grey, big_width-8, big_height-8);

                window = SDL_CreateWindow("PPM Image", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, big_width, big_height, SDL_WINDOW_SHOWN);
                if (!window) {
                    printf("Window creation failed: %c\n", SDL_GetError());
                    RUNNING = false;
                }

                renderer = SDL_CreateRenderer(window, -1, /*0*/SDL_RENDERER_ACCELERATED);
                if (!renderer) {
                    printf("Renderer creation failed: %c \n", SDL_GetError());
                    RUNNING = false;
                }

                firstImg = false;

            }
            
            texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STATIC, big_width, big_height);
            if (!texture)
            {
                printf("Texture creation failed: %c \n", SDL_GetError());
                RUNNING = false;
            }
            SDL_UpdateTexture(texture, nullptr, big_img_nn, big_width*3);
            SDL_RenderCopy(renderer, texture, nullptr, nullptr);

            fps_msg = TTF_RenderText_Solid(Sans, fps_str, White);
            fps_txt = SDL_CreateTextureFromSurface(renderer, fps_msg);

            SDL_RenderCopy(renderer, fps_txt, NULL, &Message_rect);

            SDL_RenderPresent(renderer);

            SDL_PollEvent(&event);
            SDL_DestroyTexture(texture);

            SDL_FreeSurface(fps_msg);
            SDL_DestroyTexture(fps_txt);

            free(img); free(big_img_nn); free(big_img_nn_grey); free(big_img_bic); free(big_img_bic_grey); free(big_img_dif); free(big_img_dif_grey);
            count++;
            current_img++;

            if (current_img > max_image)
                current_img = 1;
        }


        SDL_DestroyRenderer(renderer);
        SDL_DestroyWindow(window);
        SDL_Quit();

        free(width); free(height);  
    }

    catch (const std::exception& e)
    {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}

int naiveCudaExecution()
{
    int* width;
    int* height;

    int const_width;
    int const_height;

    unsigned char* img;

    int big_width;
    int big_height;

    float diff;
    unsigned char* big_img_nn;
    unsigned char* big_img_bic;
    unsigned char* big_img_dif;

    unsigned char* img_cuda;
    unsigned char* big_img_nn_cuda;
    unsigned char* big_img_bic_cuda;
    unsigned char* big_img_nn_grey_cuda;
    unsigned char* big_img_bic_grey_cuda;

    int block_dim = 16; //The x and y axis size for the block is 16 threads. Total 256 threads
    int window_size = 8;
    int scale = 2;

    bool RUNNING = true;
    bool firstImg = true;
    SDL_Window* window;
    SDL_Renderer* renderer;
    SDL_Texture* texture;
    SDL_Event event;
    SDL_PollEvent(&event);

    try
    {
        width = (int*)malloc(sizeof(int));
        height = (int*)malloc(sizeof(int));

        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        }

        if (SDL_Init(SDL_INIT_VIDEO) < 0) {
            printf("SDL initialization failed: %c\n", SDL_GetError());
            return 1;
        }

        // Initialize SDL_ttf
        if (TTF_Init() < 0) {
            printf("SDL_ttf could not initialize! TTF_Error: %s\n", TTF_GetError());
            SDL_Quit();
            return EXIT_FAILURE;
        }

        TTF_Font* Sans = TTF_OpenFont("Sans.ttf", 24);
        SDL_Color White = { 255, 255, 255 };

        char fps_str[50];
        char file_name[50];

        // as TTF_RenderText_Solid could only be used on
        // SDL_Surface then you have to create the surface first
        SDL_Surface* fps_msg;
        SDL_Texture* fps_txt;

        SDL_Rect Message_rect; //create a rect
        Message_rect.x = 5;  //controls the rect's x coordinate 
        Message_rect.y = 5; // controls the rect's y coordinte
        Message_rect.w = 200; // controls the width of the rect
        Message_rect.h = 30; // controls the height of the rect
        int count = 0;

        double frame_cap = 10;
        sprintf(fps_str, "FPS:%.*f", 3, 0.0);

        int max_image = 200;
        int current_img = 1;

        double processing_time = 0;

        while (RUNNING && event.type != SDL_QUIT)
        {
            if (count == frame_cap)
            {
                diff = 1000 * frame_cap / processing_time;
                sprintf(fps_str, "FPS:%.*f", 3, diff);

                count = 0;
                processing_time = 0;
            }

            sprintf(file_name, "./LM_Frame/image%d.ppm", current_img);

            img = (unsigned char*)readPPM(file_name, width, height);

            auto start = std::chrono::high_resolution_clock::now();

            const_width = *width;
            const_height = *height;

            big_width = const_width * scale; big_height = const_height * scale;
            big_img_nn = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height * 3);
            big_img_bic = (unsigned char*)malloc(sizeof(unsigned char) * big_width * big_height * 3);

            hipDeviceSynchronize();

            cudaStatus = hipMalloc((void**)&big_img_nn_cuda, big_width * big_height * sizeof(unsigned char) * 3);
            if (cudaStatus != hipSuccess)
                fprintf(stderr, "NN Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            cudaStatus = hipMalloc((void**)&big_img_bic_cuda, big_width * big_height * sizeof(unsigned char) * 3);
            if (cudaStatus != hipSuccess)
                fprintf(stderr, "BIC Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            cudaStatus = hipMalloc((void**)&big_img_nn_grey_cuda, big_width * big_height * sizeof(unsigned char));
            if (cudaStatus != hipSuccess)
                fprintf(stderr, "BIC Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            cudaStatus = hipMalloc((void**)&big_img_bic_grey_cuda, big_width * big_height * sizeof(unsigned char));
            if (cudaStatus != hipSuccess)
                fprintf(stderr, "BIC Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            if (hipMalloc((void**)&img_cuda, const_width * const_height * sizeof(unsigned char) * 3) != hipSuccess)
                printf("Small Image Failed To Copy To Device.\n");      //Notify failure

            hipMemcpy(img_cuda, img, sizeof(unsigned char) * const_width * const_height * 3, hipMemcpyHostToDevice);

            dim3 Grid(((big_width - 1) / block_dim) + 1, ((big_height - 1) / block_dim) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
            dim3 Block(block_dim, block_dim);

            //Launch the kernel and pass device matricies and size information
            nearestNeighborsKernel <<< Grid, Block >> > (big_img_nn_cuda, img_cuda, big_width, big_height, const_width, const_height, scale);
            bicubicInterpolationKernel <<< Grid, Block >> > (big_img_nn_cuda, img_cuda, big_width, big_height, const_width, const_height, scale);

            hipDeviceSynchronize();

            RGB2GreyscaleKernel <<< Grid, Block >>> (big_img_nn_cuda, big_img_nn_grey_cuda, big_width, big_height);
            RGB2GreyscaleKernel <<< Grid, Block >>> (big_img_bic_cuda, big_img_bic_grey_cuda, big_width, big_height);
            
            hipDeviceSynchronize();

            hipMemcpy(big_img_nn, big_img_nn_cuda, sizeof(unsigned char) * big_width * big_height * 3, hipMemcpyDeviceToHost);
            hipMemcpy(big_img_bic, big_img_bic_cuda, sizeof(unsigned char) * big_width * big_height * 3, hipMemcpyDeviceToHost);


            auto end = std::chrono::high_resolution_clock::now();
            auto dur = end - start;

            processing_time += std::chrono::duration_cast<std::chrono::milliseconds>(dur).count();

            if (firstImg)
            {
                window = SDL_CreateWindow("PPM Image", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, big_width, big_height, SDL_WINDOW_SHOWN);
                if (!window) {
                    printf("Window creation failed: %c\n", SDL_GetError());
                    RUNNING = false;
                }

                renderer = SDL_CreateRenderer(window, -1, 0);
                if (!renderer) {
                    printf("Renderer creation failed: %c \n", SDL_GetError());
                    RUNNING = false;
                }
                firstImg = false;
            }

            texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STATIC, big_width, big_height);
            if (!texture)
            {
                printf("Texture creation failed: %c \n", SDL_GetError());
                RUNNING = false;
            }

            SDL_UpdateTexture(texture, nullptr, big_img_nn, big_width * 3);
            SDL_RenderCopy(renderer, texture, nullptr, nullptr);

            fps_msg = TTF_RenderText_Solid(Sans, fps_str, White);
            fps_txt = SDL_CreateTextureFromSurface(renderer, fps_msg);

            SDL_RenderCopy(renderer, fps_txt, NULL, &Message_rect);

            SDL_RenderPresent(renderer);

            SDL_PollEvent(&event);
            SDL_DestroyTexture(texture);

            SDL_FreeSurface(fps_msg);
            SDL_DestroyTexture(fps_txt);

            free(img); free(big_img_nn); free(big_img_bic);   //free(big_img_dif);
            hipFree(img_cuda); hipFree(big_img_nn_cuda); hipFree(big_img_bic_cuda);
            hipFree(big_img_nn_grey_cuda); hipFree(big_img_bic_grey_cuda);

            count++;
            current_img++;

            if (current_img > max_image)
                current_img = 1;
        }

        SDL_DestroyRenderer(renderer);
        SDL_DestroyWindow(window);
        SDL_Quit();

        free(width);    free(height);
    }

    catch (const std::exception& e)
    {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    hipDeviceReset();
    return 0;
}

int main()
{
    return serialExecution();
    //return naiveCudaExecution();
}